#include "hip/hip_runtime.h"
// src/world/cuda_world.cu
#include <hip/hip_runtime.h>
#include "cuda/cuda_particle.h"
#include "cuda/cuda_force_registry.h"
#include "world/cuda_world.h"

namespace cudaTachyon {

    __global__ void integrateParticlesKernel(cudaParticle* particles, int count, float dt) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= count) return;
        particles[i].integrate(dt);
    }

    void runCudaWorld(std::vector<cudaParticle>& particles, std::vector<ForceEntry>& registry, float dt) {
        // Allocate GPU memory
        cudaParticle* d_particles = nullptr;
        ForceEntry* d_registry = nullptr;

        hipMalloc(&d_particles, particles.size() * sizeof(cudaParticle));
        hipMemcpy(d_particles, particles.data(), particles.size() * sizeof(cudaParticle), hipMemcpyHostToDevice);

        hipMalloc(&d_registry, registry.size() * sizeof(ForceEntry));
        hipMemcpy(d_registry, registry.data(), registry.size() * sizeof(ForceEntry), hipMemcpyHostToDevice);

        // Apply forces
        launchApplyForcesKernel(d_particles, d_registry, static_cast<int>(registry.size()));
        hipDeviceSynchronize();

        // Integrate
        int threads = 256;
        int blocks = (particles.size() + threads - 1) / threads;
        integrateParticlesKernel<<<blocks, threads>>>(d_particles, static_cast<int>(particles.size()), dt);
        hipDeviceSynchronize();

        // Copy back to CPU
        hipMemcpy(particles.data(), d_particles, particles.size() * sizeof(cudaParticle), hipMemcpyDeviceToHost);

        // Cleanup
        hipFree(d_particles);
        hipFree(d_registry);
    }

}
